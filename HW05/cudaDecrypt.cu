#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"
__device__ unsigned int aaa(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

__device__ unsigned int bbb(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = aaa(aExpb, z, p);
    z = aaa(z, z, p);
    b /= 2;
  }
  return aExpb;
}
__global__ void ccc(unsigned int *p, unsigned int *g, unsigned int *h, unsigned int *x){     
  int threadid = threadIdx.x; //thread number
  int blockid = blockIdx.x; //block number
  int Nblock = blockDim.x;  //number of threads in a block

  int id = threadid + blockid*Nblock;

  if (id <*p-1){ 

  if (bbb(*g,id+1,*p)==*h) {
        printf("Secret key found! x = %u \n", id+1);
        *x = id +1;
      }
 }
}







int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

 igned int n, p, g, h, x;
  unsigned int Nints;

  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  FILE *mess;
  mess = fopen("message.txt","r");
  FILE *key;
  key = fopen("public_key.txt","r");
  fscanf(key,"%u%u%u%u", &n, &p, &g, &h);
  fscanf(mess, "%u\n", &Nints);
  unsigned int *Zmessage =
      (unsigned int *) malloc(Nints*sizeof(unsigned int));

  unsigned int *a =
      (unsigned int *) malloc(Nints*sizeof(unsigned int));
  for (unsigned int i=0;i<Nints;i++) {
    fscanf(mess,"%u %u\n", &Zmessage[i], &a[i]);
  }

  fclose(mess);
  fclose(key);


  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    

    unsigned int *pp, *gg, *hh, *xx;
    hipMalloc(&pp, sizeof(unsigned int));
    hipMalloc(&gg, sizeof(unsigned int));
    hipMalloc(&hh, sizeof(unsigned int));
    hipMalloc(&xx, sizeof(unsigned int));

    hipMemcpy(pp, &p, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(gg, &g, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(hh, &h, sizeof(unsigned int), hipMemcpyHostToDevice);

    ccc<<<(p+1022)/1024, 1024>>>(pp, gg, hh, xx);
 
    hipMemcpy(&x, xx, sizeof(unsigned int), hipMemcpyDeviceToHost);
 }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }

  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
  ElGamalDecrypt(Zmessage,a,Nints,p,x);
  unsigned int cpi = (n-1)/8;
  convertZToString(Zmessage, Nints, message, Nints*cpi);


/* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  return 0;
}
